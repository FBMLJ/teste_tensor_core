//////////////////////////////////////////////////////////////////////
// A simple example to show how CUDA WMMA API works with Tensor Cores
//    Created by Zong-Sheng Wang @ 2018/11/25
// Performance Tips:
//    To minimize bank conflicts, you should try to shift row or 
// column of matrics in shared memory
// cmd: 
//    $ nvcc -o main main.cu -arch sm_75

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h>




#define WARP_SIZE 32

// MMA matrix tile dimensions.
#define M 16
#define N 16
#define K 16

// GEMM configuration.
#define M_TILES 256
#define N_TILES 256
#define K_TILES 256

#define M_TOTAL (M * M_TILES)
#define N_TOTAL (N * N_TILES)
#define K_TOTAL (K * K_TILES)


using namespace nvcuda;

__host__ void InitMatrix(half *A, half *B, float *C)
{
	for (int i = 0; i < M_TOTAL*K_TOTAL; i++)
		A[i] = __float2half(rand() % 1000 / 1000.0f);
	for (int i = 0; i < K_TOTAL*N_TOTAL; i++)
		B[i] = __float2half(rand() % 1000 / 1000.0f);
	for (int i = 0; i < M_TOTAL*N_TOTAL; i++)
		C[i] = rand() % 1000 / 1000.0f;
}



__global__ void WMMAF16TensorCore(half *A, half *B, float *C, float *D)
{
	int ix = (blockIdx.x * blockDim.x + threadIdx.x)/WARP_SIZE;
	int iy = (blockIdx.y * blockDim.y + threadIdx.y);
	
	wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major> b_frag;
	wmma::fragment<wmma::accumulator, M, N, K, float> ab_frag;
	wmma::fragment<wmma::accumulator, M, N, K, float> c_frag;
	
	wmma::fill_fragment(ab_frag, 0.0f);

	// AB = A*B
	int a_col, a_row, b_col, b_row, c_col, c_row;
	a_row = ix * M;
	b_row = iy * N;
	for (int k=0; k<K_TOTAL; k+=K) {
		a_col = b_col = k;

		if (a_row < M_TOTAL && a_col < K_TOTAL && b_row < K_TOTAL && b_col < N_TOTAL) {
			// Load the inputs
			wmma::load_matrix_sync(a_frag, A + a_col + a_row * M_TOTAL, M_TOTAL);
			wmma::load_matrix_sync(b_frag, B + b_col + b_col * K_TOTAL, K_TOTAL);

			// Perform the matrix multiplication
			wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
		}
	}

	// D = AB + C
	c_col = b_row;
	c_row = a_row;
	if (c_row < M_TOTAL && c_col < N_TOTAL) {
		wmma::load_matrix_sync(c_frag, C + c_col + c_row * N_TOTAL, N_TOTAL, wmma::mem_row_major);

		for (int i = 0; i < c_frag.num_elements; i++) {
			c_frag.x[i] = ab_frag.x[i] + c_frag.x[i];
		}

		// Store the output
		wmma::store_matrix_sync(D + c_col + c_row * N_TOTAL, c_frag, N_TOTAL, wmma::mem_row_major);
	}
}

hipError_t CalcWMMA(half *A, half *B, float *C, float *D)
{
	hipError_t cuda_status;
	dim3 gridDim, blockDim;
	// 16 warps in one block
	blockDim.x = 4 * WARP_SIZE; 
	blockDim.y = 4;

	gridDim.x = (M_TOTAL + (M * blockDim.x / WARP_SIZE - 1)) / (M * blockDim.x / WARP_SIZE);
	gridDim.y = (N_TOTAL + N * blockDim.y - 1) / (N * blockDim.y);

	// for Performance Metrics
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	WMMAF16TensorCore<<<gridDim, blockDim>>>(A, B, C, D);
	cuda_status = hipDeviceSynchronize();
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	// for Performance Metrics
	printf("[+] GPU(with Tensor Cores) Elapsed Time: %f ms\n", milliseconds);
	// references from https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
	printf("[+] TFLOPS: %.2f\n", ((double)M_TOTAL * N_TOTAL* K_TOTAL * 2) / milliseconds / 1e9);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return cuda_status;
}


int main()
{
	hipError_t cuda_status;
	cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		printf("hipSetDevice failed! ");
		return 1;
	}


	// Matrix on device
	half *A;
	half *B;
	float *C;
	float *D;

	// CUDA Unified Memory 
	hipMallocManaged((void **)&A, sizeof(half) * M_TOTAL * K_TOTAL);
	hipMallocManaged((void **)&B, sizeof(half) * K_TOTAL * N_TOTAL);
	hipMallocManaged((void **)&C, sizeof(float) * M_TOTAL * N_TOTAL);
	hipMallocManaged((void **)&D, sizeof(float) * M_TOTAL * N_TOTAL);
	
	// Init matrix A B C on host
	//InitHostMatrix(host_A, host_B, host_C);
	printf("[*] Initializing Matrix...\n");
	InitMatrix(A, B, C);
	printf("[+]   A: %d x %d\n", M_TOTAL, K_TOTAL);
	printf("[+]   B: %d x %d\n", K_TOTAL, N_TOTAL);
	printf("[+]   C: %d x %d\n", M_TOTAL, N_TOTAL);
	
	// computing gemm using tensor core
	printf("[*] Computing D = A * B +C with Tensor Cores...\n");
	// D = A * B +C, D holds the result after ret
	cuda_status = CalcWMMA(A, B, C, D);
	
	cuda_status = hipDeviceReset();
	if (cuda_status != hipSuccess) {
		printf("hipDeviceReset failed! ");
		return 1;
	}
	// Todo: Add a function to verify the result by using the result of CPU version implementation.

	hipFree(A);
	hipFree(B);
	hipFree(C);
	hipFree(D);

	return 0;
}