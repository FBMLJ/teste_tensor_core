#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h>

#include "hip/hip_runtime.h"
#include ""



__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

using namespace nvcuda;


half * to_half(float *vetor, int tamanho){

   float *cuda_vet;
   half *half_vet;
   hipMalloc(&cuda_vet, sizeof(float) * tamanho);
   hipMalloc(&half_vet, sizeof(float) * tamanho);
   convertFp32ToFp16<<<(tamanho)/32 + 1,32>>>(half_vet, cuda_vet, tamanho);
   
   hipFree(cuda_vet);

   hipFree(half_vet);

}

int main(){
   float vetor[10] = {1,1,1,1,1,1,1,1,1,1};
   to_half()
   
}