
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <mma.h>





#define ROWS 16*3
#define COLS 16*3



__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

using namespace nvcuda;


half * to_half(float *vetor, int tamanho){
   float *cuda_vet;
   half *half_vet;
   hipMalloc(&cuda_vet,  tamanho);
   hipMemcpy(cuda_vet, vetor, tamanho, hipMemcpyHostToDevice);
   free(vetor);
   hipMalloc(&half_vet,  tamanho);
   convertFp32ToFp16<<<(tamanho)/32 + 1,32>>>(half_vet, cuda_vet, tamanho);
   
   hipFree(cuda_vet);

  
   return half_vet;

}

int main(){
   float *vetor;
   int tam = ROWS * COLS * sizeof(float);
   vetor =(float*) malloc(tam);
   for(int i = 0 ; i < 100; i++) vetor[i] = 1;
   half *half_vet1 = to_half(vetor, tam);

   vetor =(float*) malloc(tam);
   for(int i = 0 ; i < 100; i++) vetor[i] = 1;
   half *half_vet2 = to_half(vetor, tam);


   hipFree(half_vet1);
   hipFree(half_vet2);
   
}